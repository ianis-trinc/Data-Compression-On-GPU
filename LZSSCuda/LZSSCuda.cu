#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <vector>

// Definirea funcției `min` pentru utilizare în kernel-uri CUDA
__device__ int Min(int a, int b) {
    return (a < b) ? a : b;
}
// Kernel pentru compresie LZSS
__device__ int findLongestMatch(const unsigned char* window, int windowSize, const unsigned char* lookahead, int lookaheadSize, int& matchDistance) {
    int maxLength = 0;
    for (int i = 0; i < windowSize; ++i) {
        int length = 0;
        while (length < lookaheadSize && window[i + length] == lookahead[length]) {
            ++length;
        }
        if (length > maxLength) {
            maxLength = length;
            matchDistance = windowSize - i;
        }
    }
    return maxLength;
}

__global__ void CompressKernel(const unsigned char* input, int inputLength, unsigned char* output, int* outputLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = tid * 32;
    if (offset >= inputLength) return;

    int windowSize = 4096;
    int lookaheadSize = 18;
    int minMatchLength = 3;
    int compressedIndex = offset;

    for (int i = offset; i < inputLength && i < offset + 32; ) {
        int matchLength = 0;
        int matchDistance = 0;

        if (i >= windowSize) {
            matchLength = findLongestMatch(input + i - windowSize, windowSize, input + i, Min(lookaheadSize, inputLength - i), matchDistance);
        }

        if (matchLength >= minMatchLength) {
            output[compressedIndex++] = 1; // Match flag
            output[compressedIndex++] = matchDistance >> 8;
            output[compressedIndex++] = matchDistance & 0xFF;
            output[compressedIndex++] = matchLength;
            i += matchLength;
        }
        else {
            output[compressedIndex++] = 0; // Literal flag
            output[compressedIndex++] = input[i++];
        }
    }

    outputLength[tid] = compressedIndex - offset;
}

__global__ void DecompressKernel(const unsigned char* input, int inputLength, unsigned char* output, int* outputLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = tid * 32;
    if (offset >= inputLength) return;

    int decompressedIndex = offset;

    for (int i = offset; i < inputLength && i < offset + 32; ) {
        unsigned char flag = input[i++];
        if (flag == 0) {
            output[decompressedIndex++] = input[i++];
        }
        else {
            int matchDistance = (input[i] << 8) | input[i + 1];
            int matchLength = input[i + 2];
            i += 3;
            int start = decompressedIndex - matchDistance;
            for (int j = 0; j < matchLength; ++j) {
                output[decompressedIndex++] = output[start + j];
            }
        }
    }

    outputLength[tid] = decompressedIndex - offset;
}

extern "C" {
    void CompressCuda(const unsigned char* input, int inputLength, unsigned char* output, int* outputLength) {
        unsigned char* d_input;
        unsigned char* d_output;
        int* d_outputLength;

        int numBlocks = (inputLength + 31) / 32;

        hipMalloc((void**)&d_input, inputLength);
        hipMalloc((void**)&d_output, inputLength * 2); // Dimensiunea maximă estimată pentru output
        hipMalloc((void**)&d_outputLength, numBlocks * sizeof(int));

        hipMemcpy(d_input, input, inputLength, hipMemcpyHostToDevice);

        CompressKernel << <numBlocks, 32 >> > (d_input, inputLength, d_output, d_outputLength);

        std::vector<int> h_outputLength(numBlocks);
        hipMemcpy(h_outputLength.data(), d_outputLength, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

        int totalLength = 0;
        for (int len : h_outputLength) {
            totalLength += len;
        }

        hipMemcpy(output, d_output, totalLength, hipMemcpyDeviceToHost);
        *outputLength = totalLength;

        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_outputLength);
    }

    void DecompressCuda(const unsigned char* input, int inputLength, unsigned char* output, int* outputLength) {
        unsigned char* d_input;
        unsigned char* d_output;
        int* d_outputLength;

        int numBlocks = (inputLength + 31) / 32;

        hipMalloc((void**)&d_input, inputLength);
        hipMalloc((void**)&d_output, inputLength * 2); // Dimensiunea maximă estimată pentru output
        hipMalloc((void**)&d_outputLength, numBlocks * sizeof(int));

        hipMemcpy(d_input, input, inputLength, hipMemcpyHostToDevice);

        DecompressKernel << <numBlocks, 32 >> > (d_input, inputLength, d_output, d_outputLength);

        std::vector<int> h_outputLength(numBlocks);
        hipMemcpy(h_outputLength.data(), d_outputLength, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

        int totalLength = 0;
        for (int len : h_outputLength) {
            totalLength += len;
        }

        hipMemcpy(output, d_output, totalLength, hipMemcpyDeviceToHost);
        *outputLength = totalLength;

        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_outputLength);
    }
}
